
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <iostream>
#include <chrono>


//======================
#define DEV_NO 0
#define debug 0
#define BS 64
#define half_BS BS/2
// cudaDeviceProp prop;

const int INF = ((1 << 30) - 1);
// const int V = 50010;
void input(char* inFileName);
void output(char* outFileName);
void block_FW(int B);
int ceil(int a, int b);

__device__ void assign_to_share_mem(int share[BS][BS], int* dist, int offset, int n, int x, int y);
__device__ void assign_to_device(int share[BS][BS], int* dist, int offset, int n, int x, int y);
__global__ void cal1(int *dist, int n, int Round, int B);
__global__ void cal2(int *dist, int n, int Round, int B);
__global__ void cal3(int *dist, int n, int Round, int B);

int n, m, origin_n;
// static int Dist[V][V];
int* Dist = NULL;


void display(void)
{
    printf("==========\n");
    for (int i = 0; i < origin_n; ++i) {
        for (int j = 0; j < origin_n; ++j) {
            if (Dist[i*n+j] >= INF) printf(" -1 ");
            else printf("%3d ", Dist[i*n+j]);
        }
        printf("\n");
    }
    printf("==========\n");
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&origin_n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    //padding
    n = origin_n + (BS-(origin_n%BS));
    Dist = (int*) malloc(sizeof(int)*n*n);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i*n+j] = 0;
            } else {
                Dist[i*n+j] = INF;
            }
        }
    }

    int* buffer = (int*)malloc(m*3*sizeof(int));
    fread(buffer, sizeof(int), m*3, file);

    #pragma omp parallel for schedule(static)
    for (int i=0; i<m; i++){
        Dist[buffer[i*3]*n+buffer[i*3+1]] = buffer[i*3+2];
    }
    

    if(debug) display();
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < origin_n; ++i) {
        for (int j = 0; j < origin_n; ++j) {
            if (Dist[i*n+j] >= INF) Dist[i*n+j] = INF;
        }
        fwrite(&Dist[i*n], sizeof(int), origin_n, outfile);
    }
    if(debug) display();
    fclose(outfile);

}

int ceil(int a, int b) { return (a + b - 1) / b; }

void block_FW(int B) {

    int* dist = NULL;  //device
    unsigned long Size = n * n * sizeof(int);

    hipMalloc(&dist, Size);
	hipMemcpy(dist, Dist, Size, hipMemcpyHostToDevice);

    int round = ceil(n, BS);
    dim3 grid(round, round); // num of blocks for phase3
    dim3 grid_phase2(round, 2);
    dim3 blk(half_BS,half_BS); //1024 threads for phase1

    for (int r = 0; r < round; ++r) {

        /* Phase 1*/
        cal1<<<1, blk>>>(dist, n, r, BS);

        /* Phase 2*/
        cal2<<<grid_phase2, blk>>>(dist, n, r, BS);

        // /* Phase 3*/
        cal3<<<grid, blk>>>(dist, n, r, BS);
    }
    
    hipMemcpy(Dist, dist, Size, hipMemcpyDeviceToHost);
	hipFree(dist);
}

__device__ void assign_to_share_mem(int share[BS][BS], int* dist, int offset, int n, int x, int y)
{
    share[y][x] = dist[offset + y*n + x];
    share[y+half_BS][x] = dist[offset + (y+half_BS)*n + x];
    share[y][x+half_BS] = dist[offset + y*n + x + half_BS];
    share[y+half_BS][x+half_BS] = dist[offset + (y+half_BS)*n + x + half_BS];
}

__device__ void assign_to_device(int share[BS][BS], int* dist, int offset, int n, int x, int y)
{
    dist[offset + y*n + x] = share[y][x];
    dist[offset + (y+half_BS)*n + x] = share[y+half_BS][x];
    dist[offset + y*n + x + half_BS] = share[y][x+half_BS];
    dist[offset + (y+half_BS)*n + x + half_BS] = share[y+half_BS][x+half_BS];
}

__global__ void cal1(int* dist, int n, int Round, int B)
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    int offset = BS*Round;

    
    //create shared memory
    __shared__ int Pivot[BS][BS];

    // every threads need to compute 4 points
    assign_to_share_mem(Pivot, dist, offset*(n+1), n, x, y);

    __syncthreads();

    for (int k=0; k<B; k++){
        Pivot[y][x] = min(Pivot[y][k] + Pivot[k][x], Pivot[y][x]);
        Pivot[y+half_BS][x] = min(Pivot[y+half_BS][k] + Pivot[k][x], Pivot[y+half_BS][x]);
        Pivot[y][x+half_BS] = min(Pivot[y][k] + Pivot[k][x+half_BS], Pivot[y][x+half_BS]);
        Pivot[y+half_BS][x+half_BS] = min(Pivot[y+half_BS][k] + Pivot[k][x+half_BS], Pivot[y+half_BS][x+half_BS]);
        __syncthreads(); 
    }
    
    assign_to_device(Pivot, dist, offset*(n+1), n, x, y);
}

__global__ void cal2(int* dist, int n, int Round, int B)
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    int offset = BS*Round;
    int blk_x = blockIdx.x;
    int blk_y = blockIdx.y;
    if(blk_x==Round) return;



    //create shared memory
    __shared__ int D[BS][BS];
    __shared__ int Pivot[BS][BS];

    // D depends on itself and Pivot
    if(blk_y==0){
        assign_to_share_mem(D, dist, offset*n + blk_x*BS, n, x, y);
    }
    else{
        assign_to_share_mem(D, dist, blk_x*BS*n + offset, n, x, y);
    }

    assign_to_share_mem(Pivot, dist, offset*(n+1), n, x, y);
    
    
    __syncthreads();

    
    if(blk_y==0){
        #pragma unroll 32
        for (int k=0; k<B; k++){
            D[y][x] = min(Pivot[y][k] + D[k][x], D[y][x]);
            D[y+half_BS][x] = min(Pivot[y+half_BS][k] + D[k][x], D[y+half_BS][x]);
            D[y][x+half_BS] = min(Pivot[y][k] + D[k][x+half_BS], D[y][x+half_BS]);
            D[y+half_BS][x+half_BS] = min(Pivot[y+half_BS][k] + D[k][x+half_BS], D[y+half_BS][x+half_BS]);
            __syncthreads();
        }
    }
    else{
        #pragma unroll 32
        for (int k=0; k<B; k++){
            D[y][x] = min(D[y][k] + Pivot[k][x], D[y][x]);
            D[y+half_BS][x] = min(D[y+half_BS][k] + Pivot[k][x], D[y+half_BS][x]);
            D[y][x+half_BS] = min(D[y][k] + Pivot[k][x+half_BS], D[y][x+half_BS]);
            D[y+half_BS][x+half_BS] = min(D[y+half_BS][k] + Pivot[k][x+half_BS], D[y+half_BS][x+half_BS]);
            __syncthreads();
        }
    }

    if(blk_y==0){
        assign_to_device(D, dist, offset*n + blk_x*BS, n, x, y);
    }
    else{
        assign_to_device(D, dist, blk_x*BS*n + offset, n, x, y);
    }
}

__global__ void cal3(int* dist, int n, int Round, int B)
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    int blk_x = blockIdx.x;
    int blk_y = blockIdx.y;
    int offset = BS*Round;
    if(blk_x==Round || blk_y==Round) return;

    //create shared memory
    // D depends on itself and Col&Row
    __shared__ int Col[BS][BS];
    __shared__ int Row[BS][BS];
    __shared__ int D[BS][BS];


    assign_to_share_mem(D, dist, blk_y*BS*n + blk_x*BS, n, x, y);
    assign_to_share_mem(Col, dist, blk_y*BS*n + offset, n, x, y);
    assign_to_share_mem(Row, dist, offset*n + blk_x*BS, n, x, y);

    __syncthreads();

    #pragma unroll 32
    for (int k=0; k<B; k++){
        D[y][x] = min(Col[y][k] + Row[k][x], D[y][x]);
        D[y+half_BS][x] = min(Col[y+half_BS][k] + Row[k][x], D[y+half_BS][x]);
        D[y][x+half_BS] = min(Col[y][k] + Row[k][x+half_BS], D[y][x+half_BS]);
        D[y+half_BS][x+half_BS] = min(Col[y+half_BS][k] + Row[k][x+half_BS], D[y+half_BS][x+half_BS]);
    }

    assign_to_device(D, dist, blk_y*BS*n + blk_x*BS, n, x, y);
    
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    block_FW(BS);
    output(argv[2]);
    return 0;
}
